#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;

//*******************************************

// Write down the kernels here


__device__ int round_count;

__global__ void health_initialization(int* health_device, int health)
{

    health_device[threadIdx.x] = health;
    if (threadIdx.x == 0)
    {
        round_count = 1;
    }
}

__global__ void closest_Tank_hit(int* x_device, int* y_device, int* health_device, int T, unsigned long long int* hitList)
{



    long long int Y_axis = y_device[(blockIdx.x + round_count) % T] - y_device[blockIdx.x];
    long long int X_axis = x_device[(blockIdx.x + round_count) % T] - x_device[blockIdx.x];


    if (threadIdx.x == 0)
    {
        hitList[blockIdx.x] = LLONG_MAX;
    }
    __syncthreads();

    if (threadIdx.x == blockIdx.x || health_device[threadIdx.x] <= 0 || health_device[blockIdx.x] <= 0)
    {
        return;
    }

    long long int Initial_X = x_device[threadIdx.x] - x_device[blockIdx.x];
    long long int Initial_Y = y_device[threadIdx.x] - y_device[blockIdx.x];
    int tank_count = 0;

    if (X_axis * Initial_Y == Y_axis * Initial_X && X_axis * Initial_X >= 0 && Y_axis * Initial_Y >= 0)
    {
        unsigned long long temp_dist = ((unsigned long long int)(abs(Initial_Y)) + ((unsigned long long int)(abs(Initial_X))) << 32);
        tank_count++;
        unsigned long long temp_valX = temp_dist | ((unsigned long long)threadIdx.x);
        atomicMin(&hitList[blockIdx.x], temp_valX);
    }
}

__global__ void score_updating(int* device_health, int* scoreList, int* over, int T, unsigned long long* target_arr)
{

    if (threadIdx.x == 0)
    {
        round_count++;
        if (round_count % T == 0)
        {
            round_count = 1;
        }
    }
    int a = 1;
    if (target_arr[threadIdx.x] != (LLONG_MAX)) {
        int tar_tank = int(target_arr[threadIdx.x] & (LLONG_MAX));

        int previous_hth = atomicSub(&device_health[tar_tank], a);

        if (previous_hth == a)
        {
            atomicAdd(over, a);
        }
        scoreList[threadIdx.x] = scoreList[threadIdx.x] + a;
    }
}

//***********************************************


int main(int argc, char** argv)
{
    // Variable declarations
    int M, N, T, H, * xcoord, * ycoord, * score;


    FILE* inputfilepointer;

    //File Opening for read
    char* inputfilename = argv[1];
    inputfilepointer = fopen(inputfilename, "r");

    if (inputfilepointer == NULL) {
        printf("input.txt file failed to open.");
        return 0;
    }

    fscanf(inputfilepointer, "%d", &M);
    fscanf(inputfilepointer, "%d", &N);
    fscanf(inputfilepointer, "%d", &T); // T is number of Tanks
    fscanf(inputfilepointer, "%d", &H); // H is the starting Health point of each Tank

    // Allocate memory on CPU
    xcoord = (int*)malloc(T * sizeof(int));  // X coordinate of each tank
    ycoord = (int*)malloc(T * sizeof(int));  // Y coordinate of each tank
    score = (int*)malloc(T * sizeof(int));  // Score of each tank (ensure that at the end you have copied back the score calculations on the GPU back to this allocation)

    // Get the Input of Tank coordinates
    for (int i = 0; i < T; i++)
    {
        fscanf(inputfilepointer, "%d", &xcoord[i]);
        fscanf(inputfilepointer, "%d", &ycoord[i]);
    }


    auto start = chrono::high_resolution_clock::now();

    //*********************************
    // Your Code begins here (Do not change anything in main() above this comment)
    //********************************

    int* Score_device, * health_device;
    hipMalloc(&health_device, T * sizeof(int));
    hipMalloc(&Score_device, T * sizeof(int));
    hipMemset(health_device, 0, T * sizeof(int));
    hipMemset(Score_device, 0, T * sizeof(int));




    health_initialization << <1, T >> > (health_device, H);
    hipDeviceSynchronize();

    unsigned long long* device_hitArr;
    hipMalloc(&device_hitArr, T * sizeof(unsigned long long int));
    int host_over_value = 0;

    int* x_device, * device_over_value, * y_device;
    hipMalloc(&device_over_value, 1 * sizeof(int));
    hipMalloc(&x_device, T * sizeof(int));
    hipMalloc(&y_device, T * sizeof(int));
    hipMemcpy(x_device, xcoord, T * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(device_over_value, 0, 1 * sizeof(int));
    hipMemcpy(y_device, ycoord, T * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();

    int blocksize = T, threadsperblock = T;
    int check = 0;
    while (1)
    {
        if (host_over_value < T - 1) {
            closest_Tank_hit << <blocksize, threadsperblock >> > (x_device, y_device, health_device, T, device_hitArr);
            hipDeviceSynchronize();
            check++;
            score_updating << <1, threadsperblock >> > (health_device, Score_device, device_over_value, T, device_hitArr);
            hipDeviceSynchronize();
            check--;
            hipMemcpy(&host_over_value, device_over_value, sizeof(int), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
        }
        else
            break;
    }

    hipMemcpy(score, Score_device, T * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(y_device);
    hipFree(x_device);
    hipFree(health_device);
    
    

    //*********************************
    // Your Code ends here (Do not change anything in main() below this comment)
    //********************************

    auto end = chrono::high_resolution_clock::now();

    chrono::duration<double, std::micro> timeTaken = end - start;

    printf("Execution time : %f\n", timeTaken.count());

    // Output
    char* outputfilename = argv[2];
    char* exectimefilename = argv[3];
    FILE* outputfilepointer;
    outputfilepointer = fopen(outputfilename, "w");

    for (int i = 0; i < T; i++)
    {
        fprintf(outputfilepointer, "%d\n", score[i]);
    }
    fclose(inputfilepointer);
    fclose(outputfilepointer);

    outputfilepointer = fopen(exectimefilename, "w");
    fprintf(outputfilepointer, "%f", timeTaken.count());
    fclose(outputfilepointer);

    free(xcoord);
    free(ycoord);
    free(score);
    hipDeviceSynchronize();
    return 0;
}
